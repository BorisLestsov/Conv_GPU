/*
 * functions.cu
 *
 *  Created on: Jan 4, 2017
 *      Author: boris
 */

#include "functions.cuh"

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

