#include "hip/hip_runtime.h"
#include "Filters.h"
#include "functions.cuh"

#include <hip/hip_runtime.h>

UnnormalizedFilter::UnnormalizedFilter(
		const FilterKernel & kernel_p,
		bool check_range):

        kernel (kernel_p),
        hor_radius ( kernel_p.n_cols / 2),
		vert_radius ( kernel_p.n_rows / 2),
        check(check_range)
{
		_conv_function = &UnnormalizedFilter::_conv_CPU;
}


Pixel UnnormalizedFilter::_conv_GPU(const Image &m) const {
	return make_tuple(0, 0, 0);
}

Pixel UnnormalizedFilter::_conv_CPU(const Image &m) const {
	double r, g, b, sum_r = 0, sum_g = 0, sum_b = 0;

	for (uint i = 0; i < kernel.n_rows; ++i) {
		for (uint j = 0; j < kernel.n_cols; ++j) {
			r = static_cast<double>(get<0>(m(i, j)));
			g = static_cast<double>(get<1>(m(i, j)));
			b = static_cast<double>(get<2>(m(i, j)));
			r = r * kernel(i, j);
			g = g * kernel(i, j);
			b = b * kernel(i, j);
			sum_r += r;
			sum_g += g;
			sum_b += b;
		}
	}
	if (check){
		if (sum_r > 255)
			sum_r = 255;
		else if (sum_r < 0)
			sum_r = 0;

		if (sum_g > 255)
			sum_g = 255;
		else if (sum_g < 0)
			sum_g = 0;

		if (sum_b > 255)
			sum_b = 255;
		else if (sum_b < 0)
			sum_b = 0;
	}
	return make_tuple(sum_r, sum_g, sum_b);
}

Pixel UnnormalizedFilter::operator () (const Image &m) const {
    return (this->*_conv_function)(m);
}


Image UnnormalizedFilter::convolve(const Image& img) const {
	unsigned char* d_img, *d_res;
	unsigned char* img_raw, *img_res;
	float* d_ker;
	float* host_ker;
	unsigned int res_rows = img.n_rows-2*hor_radius;
	unsigned int res_cols = img.n_cols-2*vert_radius;
	unsigned int hor_size = hor_radius*2 + 1;
	unsigned int vert_size = vert_radius*2 + 1;

	img_raw = (unsigned char*) malloc(img.n_rows*img.n_cols*3);
	img_res = (unsigned char*) malloc(res_rows*res_cols*3);
	host_ker = (float*) malloc(hor_size*vert_size*sizeof(float));


	CUDA_CHECK_RETURN( hipMalloc(
			(void**) &d_img,
			img.n_rows*img.n_cols*3) );
	CUDA_CHECK_RETURN( hipMalloc(
			(void**) &d_res,
			res_rows*res_cols*3) );
	CUDA_CHECK_RETURN( hipMalloc(
			(void**) &d_ker,
			hor_size*vert_size*sizeof(float)) );

	for (uint i = 0; i < img.n_rows; ++i){
		for (uint j = 0; j < img.n_cols; ++j){
			img_raw[(i*img.n_cols + j)*3 + 0] = get<0>(img(i, j));
			img_raw[(i*img.n_cols + j)*3 + 1] = get<1>(img(i, j));
			img_raw[(i*img.n_cols + j)*3 + 2] = get<2>(img(i, j));
		}
	}

	for (uint i = 0; i < hor_size; ++i){
		for (uint j = 0; j < vert_size; ++j){
			host_ker[i*hor_size + j] = kernel(i,j);
		}
	}

	CUDA_CHECK_RETURN( hipMemcpy(d_img, img_raw,
			img.n_rows*img.n_cols*3,
			hipMemcpyHostToDevice)
			);
	CUDA_CHECK_RETURN( hipMemset(d_res,
				0,
				res_rows*res_cols*3)
				);
	CUDA_CHECK_RETURN( hipMemcpy(d_ker, host_ker,
				hor_size*vert_size*sizeof(float) ,
				hipMemcpyHostToDevice)
				);


	dim3 block_grid(res_rows, res_cols);
	dim3 thread_grid(hor_size, vert_size, 3);
	compute<<<block_grid, thread_grid>>>(
			res_rows,
			res_cols,
			d_img,
			d_res,
			hor_radius,
			vert_radius,
			d_ker);


	CUDA_CHECK_RETURN( hipMemcpy(img_res, d_res,
				res_rows*res_cols*3,
				hipMemcpyDeviceToHost)
				);

	Image res(res_rows,  res_cols);

	for (uint i = 0; i < res_rows; ++i){
		for (uint j = 0; j < res_cols; ++j){
			get<0>(res(i, j)) = img_res[(i*res_cols + j)*3 + 0];
			get<1>(res(i, j)) = img_res[(i*res_cols + j)*3 + 1];
			get<2>(res(i, j)) = img_res[(i*res_cols + j)*3 + 2];
		}
	}

	CUDA_CHECK_RETURN( hipFree(d_img) );
	CUDA_CHECK_RETURN( hipFree(d_res) );

	free((void*) img_raw);
	free((void*) img_res);

	return res;
}



FloatFilter::FloatFilter(const FilterKernel & kernel_p):
    kernel (kernel_p),
    hor_radius (kernel_p.n_cols / 2),
    vert_radius (kernel_p.n_rows / 2)
{}

double FloatFilter::operator () (const Image &m) const
{
    uint hor_size = 2 * hor_radius + 1;
    uint vert_size = 2 * vert_radius + 1;
    double r, sum_r = 0;

    for (uint i = 0; i < vert_size; ++i) {
        for (uint j = 0; j < hor_size; ++j) {
            r = static_cast<double>(get<0>(m(i, j)));
            r = r * kernel(i, j);
            sum_r += r;
        }
    }
    return sum_r;
}


LBPFilter::LBPFilter():
        hor_radius (1),
        vert_radius (1)
{}

unsigned char LBPFilter::operator () (const Image& m) const
{
    char num = 0;
    uint cen_pix = get<0>(m(1,1));

    if (get<0>(m(1, 2)) >= cen_pix)
        num = num | R_MASK;
    if (get<0>(m(0, 2)) >= cen_pix)
        num = num | RU_MASK;
    if (get<0>(m(0, 1)) >= cen_pix)
        num = num | U_MASK;
    if (get<0>(m(0, 0)) >= cen_pix)
        num = num | LU_MASK;
    if (get<0>(m(1, 0)) >= cen_pix)
        num = num | L_MASK;
    if (get<0>(m(2, 0)) >= cen_pix)
        num = num | LD_MASK;
    if (get<0>(m(2, 1)) >= cen_pix)
        num = num | D_MASK;
    if (get<0>(m(2, 2)) >= cen_pix)
        num = num | RD_MASK;

    return num;
}


FilterKernel make_gaussian_kernel(double sigma, int radius){
	if (radius < 0)
		throw std::invalid_argument("Wrong filter radius");

	uint size = radius * 2 + 1;
	FilterKernel gauss_kernel(size, size);

	double sum = 0.0;
	for (uint i = 0; i < size; ++i) {
		for (uint j = 0; j < size; ++j) {
			gauss_kernel(i, j) = std::exp(-0.5 * ((i - radius)*(i - radius) / (sigma*sigma)
					+ (j - radius)*(j - radius) / (sigma*sigma)))
						   / (2 * M_PI * sigma*sigma);
			sum += gauss_kernel(i, j);
		}
	}

	for (uint i = 0; i < size; ++i){
		for (uint j = 0; j < size; ++j){
			gauss_kernel(i, j) /= sum;
		}
	}
	return gauss_kernel;
}
